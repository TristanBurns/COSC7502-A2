#include "hip/hip_runtime.h"
#include "eigensolver.h"
#include "randutil.h"
#include <string>
#include <iostream>
#include <iomanip>

// global variables to store the matrix

double* M = nullptr;
double* X = nullptr;
double* Y = nullptr;
double* Ycuda = nullptr;
int N = 9;


#define VectorLength 4
#define VectorLength512 4

void checkError(hipError_t e)
{
   if (e != hipSuccess)
   {
      std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
      abort();
   }
}

__global__
void kernel(float *X, float *M, float *Y, const int N)
{
    int tid=threadIdx.x+blockIdx.x*blockDim.x; //thread id
    float sum=0;
    if(tid<N)
    {
        for(int i=0; i<N; i++)
        {
            //sum += X[i]*M[(tid*N)+i]; 
            sum += X[i]*M[(i*N)+tid];//chec
        }
        Y[tid]=sum;
    }
}



// implementation of the matrix-vector multiply function
void cudaMatrixVectorMultiply(double* Y, const double* X)
{  
    float* xFloat = new float[N];
    float* mFloat = new float[N*N];
    float* yFloat = new float[N];
   
    for (int i = 0 ; i < N; i++)
    {
        xFloat[i] = (float) X[i];
    }
    
    for (int j = 0 ; j < N; j++)
    {
        for (int i = 0 ; i < N; i++)
        {
            //mFloat[i*N+j]  = (float) M[i*N+j];
            mFloat[j*N+i]  = (float) M[i*N+j];
        }
    }

   float* xDevice;
   float* yDevice;
   float* mDevice;
   checkError(hipMalloc(&xDevice, N*sizeof(float)));
   checkError(hipMalloc(&yDevice, N*sizeof(float)));
   checkError(hipMalloc(&mDevice, N*N*sizeof(float)));

   checkError(hipMemcpy(xDevice, xFloat, sizeof(float)*N, hipMemcpyHostToDevice));
   checkError(hipMemcpy(mDevice, mFloat, sizeof(float)*N*N, hipMemcpyHostToDevice));

   int Threads = 256;
   int Blocks = (N+Threads-1)/Threads;
   kernel<<<Blocks, Threads>>>(xDevice, mDevice, yDevice, N);
   //checkError(hipDeviceSynchronize());
   checkError(hipMemcpy(yFloat, yDevice, N*sizeof(float), hipMemcpyDeviceToHost));
   

    for (int i = 0 ; i < N; i++)
    {
        Y[i]  = (double) yFloat[i];
    }
  
}

   

void MatrixVectorMultiply(double* Y, const double* X)
{
   for (int i = 0; i < N; ++i)
   {
      Y[i] = 0;
      for (int j = 0; j < N; ++j)
      {
         Y[i] += M[i*N+j] * X[j];
      }
   }
}


int main()
{
    randutil::seed(4);
    X = static_cast<double*>(malloc(N*sizeof(double)));
    
    std::cout << "X = [ " ;
    for (int i = 0; i < N; ++i)
    {
        X[i] = randutil::randn();
        std::cout << X[i] <<" "; 
    }
    std::cout << " ]"<<std::endl;
    M = static_cast<double*>(malloc(N*N*sizeof(double)));
   std::cout << "M = [ " ;
  
   for (int i = 0; i < N; ++i)
   {
       
      M[i*N+i] = std::sqrt(2.0) * randutil::randn();
      
      for (int j = 0; j < N; ++j)
      {
         M[i*N + j] = randutil::randn();
         std::cout << M[i*N + j] <<" "; 
      }
      std::cout << std::endl;
   }

    Y = static_cast<double*>(malloc(N*sizeof(double)));
    MatrixVectorMultiply(Y, X);
    std::cout << "Y = [ " ;
    for (int i = 0; i < N; ++i)
    {
        std::cout << Y[i] <<" "; 
    }
    std::cout << " ]"<<std::endl;


    Ycuda = static_cast<double*>(malloc(N*sizeof(double)));

  // allocate memory on the device

 

   cudaMatrixVectorMultiply(Ycuda,X);
   

    std::cout << "Ycuda = [ " ;
    for (int i = 0; i < N; ++i)
    {
        std::cout << Ycuda[i] <<" "; 
    }
    std::cout << " ]"<<std::endl;

    
    std::cout << "error = [ " ;
    for (int i = 0; i < N; ++i)
    {
        std::cout << Ycuda[i]-Y[i] <<" "; 
    }
    std::cout << " ]"<<std::endl;


    std::cout<<"sizeof double: "<<sizeof(double)<<std::endl;
return 0;
}